#include "hip/hip_runtime.h"
#define _LFS_LARGEFILE          1
#define _LFS64_LARGEFILE        1
#define _LFS64_STDIO		1
#define _LARGEFILE64_SOURCE    	1
#define _FILE_OFFSET_BITS 64

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define MAX_STREAMS 4
#define MAX_CARDS 2

#define BLOCK_SIZE 1014
#define MAX_THREAD_PER_BLOCK 1014

long long unsigned total_sub,total_data;
clock_t t_data1,t_data2;


unsigned long work_per_thread = 100;

#define MAX 1000

#define PATTERNSIZE 32
#define MAX_TOTAL_PATTERN 16

__constant__ char cpattern[sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN];


char *pattern_arr[MAX];
long long unsigned *count_found[MAX];

int total_pattern;
long long  unsigned total_found;
int TOTAL_THREADS_PER_BLOCK ;
int num_devices=0;
unsigned mb=0;        
int Rround=0;
int max_str;

#ifdef CHUNK2G
unsigned long long chunkSize =0x80000000; //2G
#elif CHUNK3G
unsigned long long chunkSize =0xc0000000; //3G
#elif CHUNK4G
 unsigned long long chunkSize =0x100000000; //4G
#elif CHUNK5G
unsigned long long chunkSize =0x140000000; //5G
#elif CHUNK1M
unsigned long long chunkSize =0x100000;//1M
#elif CHUNK64K
unsigned long long chunkSize =0x10000;//64KB
#elif CHUNK1G
 unsigned long long chunkSize =0x40000000; //1G
#elif CHUNK6G
unsigned long long chunkSize =0x180000000; //6G
#else
unsigned long long chunkSize =0x79999999; //1.99G default
#endif



FILE* f_b;
  

char* data_b;

texture<unsigned char, 1, hipReadModeNormalizedFloat> tpattern;

__global__ void searchb_all(char* data,  unsigned long long len_data, unsigned mb, short* pos, int p_count)
{ 
	// use constant memory cpattern
        unsigned long long mycount=0;
        //For all blocks
        unsigned long long j,i =blockIdx.x * blockDim.x + threadIdx.x;

        const int numThreads = blockDim.x * gridDim.x;
		char found,k;
		char *pattern;


   for (; i < len_data-mb+1; i+=numThreads  ) {

        for(k=0; k < p_count; k++) {
	    found=0;
	    pattern = (cpattern+k*PATTERNSIZE);	//constant memory cpattern

           if (data[i] == pattern[0]) {
	      found=1;

            for ( j=1; i+j < len_data && pattern[j] != '\0' && j<PATTERNSIZE; j++) {
             if (data[i+j] != pattern[j])     {found=0; break;}
            }


             if (found) {
                 pos[i] += 1; 
                 mycount++;
             }

          }//end if matching
       } // end for p_count

   }
}//end of Kernel


__global__ void searchb_all_shared(char* data, char* allpattern, unsigned long long len_data, unsigned mb, short* pos, int p_count)
{


        unsigned long long mycount=0;

        unsigned long long j,i =blockIdx.x * blockDim.x + threadIdx.x;

        const int numThreads = blockDim.x * gridDim.x;
        char found,k;
        char *pattern=allpattern;
	__shared__ char allpattern_s [MAX_TOTAL_PATTERN*PATTERNSIZE];

	if (threadIdx.x < MAX_TOTAL_PATTERN*PATTERNSIZE)
         	allpattern_s[threadIdx.x]= allpattern[threadIdx.x];

	 __syncthreads();

    
     for (; i < len_data-mb+1; i+=numThreads  ) {

        for(k=0; k < p_count; k++) {
            found=0;
            pattern = (allpattern_s+k*PATTERNSIZE);

           if (data[i] == pattern[0]) {
              found=1;

            for ( j=1; i+j < len_data && pattern[j] != '\0' && j<PATTERNSIZE; j++) {
             if (data[i+j] != pattern[j])     {found=0; break;}
            }


             if (found) {
                 pos[i] += 1;
                 mycount++;
             }


          }//end if matching
       } // end for p_count

   }
}//end of Kernel

 

__global__ void searchb_all_texture(char* data,   unsigned long long len_data, unsigned mb, short* pos, int p_count)
{


        unsigned long long mycount=0;
        //For all blocks
        unsigned long long j,i =blockIdx.x * blockDim.x + threadIdx.x;
        int p;
        const int numThreads = blockDim.x * gridDim.x;
        char found,k;
        char *pattern;
	__shared__ char allpattern_s [MAX_TOTAL_PATTERN*PATTERNSIZE];
		char c;

	if (i< MAX_TOTAL_PATTERN*PATTERNSIZE) //copy pattern from texture to shared
      { 	
	p=i; c= allpattern_s[i]=  tex1Dfetch(tpattern, p); 
      	
      }

	 __syncthreads();

         
      for (; i < len_data-mb+1; i+=numThreads  ) {

        for(p=0,k=0; k < p_count; k++) {
            found=0;
            p += k*PATTERNSIZE; 

            pattern = (allpattern_s+k*PATTERNSIZE);
            c=*pattern;

        // Detect the first matching character
           if (data[i] == c) {

              found=1;

        // Loop through next keyword character
              for ( j=1; i+j < len_data && c != '\0' && j<PATTERNSIZE; j++) {
       	
                c=pattern[j];	
             if (data[i+j] != c)     {found=0; break;}
            }


             if (found) {
     // Store the first matching character to the result list

                 pos[i] += 1;
                 mycount++;
             }

          }//end if matching
       } // end for p_count

   }
}//end of Kernel

 


unsigned long long count_total_found(short *arr,  unsigned long long   n)
	{
		 unsigned long long i;
		 unsigned long long c=0;
		 printf(" size %lld ",n);
		 for (i=0; i < n; i++)  {
			 //printf("i%ld arr[i] %d\n", i, arr[i]);  
			 if (arr[i] >0 ) { 
 			    c += (unsigned long long) arr[i]; //("here:");
			 }
		 }
		 return c;
}

 
int main(int argc, char** argv)
{
    printf("Running with chunksize %ld \n",chunkSize);
	int cuda_device = 0; // device ID


	unsigned long long arr_nb[MAX_CARDS];           // number of ints in the bit data set
	int j;

	int nStreams= MAX_STREAMS;
	hipStream_t stream[MAX_CARDS]; 
	 

	//start Timer
	hipError_t error;   // capture returned error code
    hipEvent_t start_event, stop_event; // data structures to capture events in GPU
     float time_main_b, total_time_main_b=0.0;
	
	// Sanity checks
	{
	    // check the compute capability of the device A

		hipGetDeviceCount(&num_devices) ;
	    if(0==num_devices)
	    {
	        printf("your system does not have a CUDA capable A device\n");
	        return 1;
	    }
    	 
	    // check if the command-line chosen device ID is within range, exit if not
	    

    	hipSetDevice( cuda_device );

		if ( argc < 3 ) {
      		printf("Usage: %s  <data_file_b> <string_substring1-..99>\n",argv[0]);
      		return -1;
    	}
	} // end of safe checks

	//Cuda Device  information

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	printDevProp(deviceProp);
	if( (deviceProp.major == 2) && (deviceProp.minor < 2)){ 
		printf("\ndevice %s does not have compute capability 2.2 or later\n",deviceProp.name);}
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, cuda_device);
	printf("***num SMs %d\n",numSMs);//print streaming multiprocessors
	

	//create stream for each device
  for (int l=0; l < num_devices; l++) {
    hipSetDevice(l);
	nStreams = MAX_STREAMS;
	error = hipStreamCreate(&stream[l]);
 
  }

  	error = hipGetLastError();
	if ( error ) { 	
		printf("Error caught-1: %s\n", hipGetErrorString( error ));
	}
		     
		
	//Open data File
	if ((f_b = fopen(argv[1] , "r")) == NULL ) { 
		printf("Error : read file %s\n",argv[1]); return 0; }
	 
 
	long double total_diff2=0.0;	
    long double total_time_data = 0.0, total_time_pat =0.0, total_time_pos=0.0;
 
   
	
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	printf("texture memory substring, chunksize %lld",chunkSize);
	// copy input substring to host substring array called pattern

	mb = 0;
 
 char *pattern= (char *) malloc(sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN);
	if (pattern == NULL) 
	    printf("error alloc whole patterns\n");
	    memset(pattern,0,sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN); 
        pattern_arr[0] =  pattern;
	for (j=2; j < argc; j++)
	 {
		   
		  pattern_arr[total_pattern] = pattern+(PATTERNSIZE*total_pattern);
		  memcpy(pattern_arr[total_pattern], argv[j], sizeof(char)*(strlen(argv[j])+1));
		  printf(" %s ",pattern_arr[total_pattern]);
		   mb= (mb > strlen(pattern_arr[total_pattern])? mb : strlen(pattern_arr[total_pattern]));
		  total_pattern++; 
		 
	 }
 	max_str = mb; // keep max string length
	printf("\n total_pattern =%d \n", total_pattern);

	
	//Device's text
	//allocate Device memory
	 
	// read in the filename and string pattern to be searched
	
	unsigned long long  alloc_size =(mb-1+chunkSize)*sizeof(char);
	char *arr_data_b[MAX_CARDS];
	unsigned long long  countc;

	short *arr_pos[MAX_CARDS];
	short *arr_d_pos[MAX_CARDS];
	char *arr_d_data_b[MAX_CARDS];
	char *arr_d_substr_b[MAX_CARDS];

	for (int l =0; l < num_devices; l++) {
	hipSetDevice(l);
	      hipHostMalloc((void**)&arr_data_b[l], alloc_size) ; // host pinned
 
	    if (arr_data_b[l] == NULL) printf("alloc data_b error \n");  
	
	
	}
	 for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
			arr_pos[l] = (short *) malloc (((mb-1)+chunkSize)*sizeof(short));
		  
			if(arr_pos[l] == NULL) {printf("error alloc pos\n"); exit(-1);}
			
	 }


	for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
	
	        hipMalloc((void**)&arr_d_pos[l],(mb-1+chunkSize)*sizeof(short));//

     	    if (arr_d_pos[l] == NULL)
	    	  printf("couldn't allocate d_pos device %d\n",l);
		 
	     hipMalloc((void**)&arr_d_data_b[l], alloc_size) ;//
	  
	    if (arr_d_data_b[l] == NULL)
		   printf("couldn't allocate d_data_b device %ld\n",l);
	      
	      // ** we use these code if we copy copy substring to device global mem
	       hipMalloc((void**)&arr_d_substr_b[l], sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN);
			//arr_d_substr_b[l] = &cpattern[l][0];
            if (arr_d_substr_b[l] == NULL)
                printf("couldn't allocate d__substr_b device %ld\n",l);

	      error = hipMemset(arr_d_substr_b[l],0,sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN);
          if ( error ) {  printf("Error caught-hipMemset d_substr_b: %s %d\n", hipGetErrorString( error ),error);}  
		//** 

		for (j=0; j < total_pattern; j++)
	           printf("copy arr_d_sub %s \n",pattern_arr[j]);
	    error= hipMemcpy(arr_d_substr_b[l],pattern_arr[0],PATTERNSIZE*MAX_TOTAL_PATTERN*sizeof(char),hipMemcpyHostToDevice);

         if ( error ) {  printf("Error caught-hipMemcpyToSymbol cpattern: %s %d device %d\n", hipGetErrorString( error ),error,l);}  

	}





	size_t cur_free, cur_total;

	printf("end memcpy arr_d_substr\n");
	for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
	
	      hipMemGetInfo(&cur_free,&cur_total); 

    	   printf("device %d: free %ld KB of total %ld KB\n",l,cur_free/1024,cur_total/1024);
	}
	 
	 
	int num_block;
	while ( !feof (f_b)) {
		num_block=0;
		for (int l =0; l < num_devices; l++) {  // looping read file chunk to devices
		  
		   countc=fread(arr_data_b[l],sizeof(char),chunkSize+mb-1,f_b);
		   if (countc <= 0 ) break;
			  
			arr_nb[l] =   ( unsigned long long ) (countc/sizeof(char));
			if (!feof(f_b)) fseeko(f_b,-((unsigned long long)mb-1),SEEK_CUR);
		  printf("read for card %d size %u \n",l, arr_nb[l]);
		  num_block++;
		}
 

	 
	   
	
	//Find 
		TOTAL_THREADS_PER_BLOCK = MAX_THREAD_PER_BLOCK ;  /**/
		
	 

		//H2D copy all data_b to all device
		t_data1= clock();

		for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
			if (l >= num_block)  break; // handle the case when  the numblock read is less than  numdevice
			hipMemcpyAsync( arr_d_data_b[l], arr_data_b[l],  arr_nb[l]*sizeof(char), hipMemcpyHostToDevice,stream[l]);
			printf("copy up data_b device %d\n",l);
		}	

		for (int l =0; l < num_devices; l++) {
		     hipSetDevice(l);
		     hipStreamSynchronize(stream[l]);
		}
		t_data2= clock();
	    long double diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
	
	    printf("\ntimeCopyH2D-1 %Lf ms ",diff2);
            Rround++;
		total_diff2 += diff2;total_time_data += diff2;

		// clear device memory for results pos
	 
			 	
		for (int l =0; l < num_devices; l++) {
			   hipSetDevice(l);
			   memset(arr_pos[l],(short) 0,arr_nb[l]*sizeof(short));  
			   hipMemset(arr_d_pos[l],(short)0,sizeof(short)*arr_nb[l]);
		}

         error = hipGetLastError();
         if ( error ) {  printf("Error caught-2- memset d_pos: %s\n", hipGetErrorString( error ));}
	     printf("after mem set pos pattern . \n" );
			 
		// stop timer
		
		
	    hipEventRecord(start_event, 0);


        for (int l =0; l < num_devices; l++) {
             hipSetDevice(l);
             printf("RunK>Dev %d\n ",l);
             if (l >= num_block)  break;

	  		error =hipBindTexture(0, tpattern, arr_d_substr_b[l], PATTERNSIZE*MAX_TOTAL_PATTERN*sizeof(char));
			if ( error ) {  printf("Error caught-1.1: %s\n", hipGetErrorString( error ));}

            searchb_all_texture<<<16*numSMs,128,0,stream[l] >>>(arr_d_data_b[l],arr_nb[l],mb,arr_d_pos[l],total_pattern);
		 
            error = hipGetLastError();
            if ( error ) {  printf("Error caught-1: %s\n", hipGetErrorString( error ));}
            hipUnbindTexture(tpattern);
         }	
		hipEventRecord(stop_event, 0);
        hipEventSynchronize( stop_event );
        //Calculate time
        hipEventElapsedTime( &time_main_b, start_event, stop_event );
	    
        error = hipGetLastError();
         if ( error ) {  printf("Error caught-1: %s\n", hipGetErrorString( error ));}

		// copy results back from pos
            for (int l =0; l < num_devices; l++) {
	             hipSetDevice(l);
	             t_data1 =clock();
	             if (l >= num_block)  break;
	             hipMemcpy(arr_pos[l], arr_d_pos[l], (arr_nb[l])*sizeof(short), hipMemcpyDeviceToHost) ;
	             error = hipGetLastError();
	             if ( error ) {  printf("Error caught-2: %s\n", hipGetErrorString( error ));}

	              t_data2= clock();
	              diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
	             printf("timeCopyD2H-3 %Lf ms ",diff2);
	             Rround++;
	             total_diff2 += diff2;
	             total_time_pos += diff2;

                //Print Time
                printf(" timeMainSearch %lf ms ", time_main_b);
                total_time_main_b += time_main_b;

                 unsigned long long  t_f= count_total_found(arr_pos[l],arr_nb[l]-mb+1);
                 printf ("nb %llu : \n ",arr_nb[l]);
                 
                printf("current_found %llu  \n", t_f);
                total_found += t_f;
                
             }

		if (feof(f_b) || countc <=0 )
		 break;
	    

		}//end while main eof
		 

		
        printf("\ntimeCopyH2D %Lf ms ",total_diff2);
		printf("time main search %lf ms ", total_time_main_b);
        printf("totalAll = %Lf ", total_diff2+total_time_main_b);
		printf("Found %lu ",total_found);
        printf("Round %d \n", Rround);

		printf("\ntotal_time_data %Lf ms total_time_pat %Lf ms " , total_time_data, total_time_pat);
        printf(" total_time_pos %Lf ms TotalH2D %Lf ms ", total_time_pos, total_time_data+total_time_pat+total_time_pos);

		

		//Free Input
		 
	for (int l =0; l < num_devices; l++) {
		hipSetDevice(l);
		
		hipFree(arr_d_data_b[l]); 
		hipFree(arr_d_pos[l]); 
		hipFree(arr_d_substr_b[l]);  
	 
	}
	hipEventDestroy( start_event ); 
	hipEventDestroy( stop_event ); 

			 
	for (int l =0; l < num_devices; l++) {
		free(arr_pos[l]);
		hipHostFree(arr_data_b[l]);
	}
 
	free(pattern);

		//Close Input File
	fclose(f_b);
	for (int l =0; l < num_devices; l++) {
		hipSetDevice(l);
	 	error = hipStreamDestroy(stream[l]);
 
	}
	printf("\nEnd");
	return 0;

}//**********************************************************************************