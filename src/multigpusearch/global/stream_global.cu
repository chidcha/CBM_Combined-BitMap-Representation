#define _LFS_LARGEFILE          1
#define _LFS64_LARGEFILE        1
#define _LFS64_STDIO                    1
#define _LARGEFILE64_SOURCE     1

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

//SEP 18-19 AUTHOR: CHANTANA CHANTRAPORNCHAI
//3_1 
//3_2
//3_3 NOO
// 3_6 stream with 2 cards
#define IMAGE_FILE_LARGE_ADDRESS_AWARE 0x0020

#define MAX_STREAMS 4
#define MAX_CARDS 2

#define BLOCK_SIZE 1014
#define MAX_THREAD_PER_BLOCK 1014
#define TOTAL_THREADS 1024
long long unsigned total_sub,total_data;
clock_t t_sub1,t_sub2,t_data1,t_data2;

const unsigned long long chunkSize =1 << 30 ;
unsigned long work_per_thread = 100;
#define MAX 100
char *pattern_arr[MAX];
int *count_found[MAX];

unsigned count_f[1];
unsigned *d_count_f;

int total_pattern;
 unsigned total_found=0;
int TOTAL_THREADS_PER_BLOCK ;
 int num_devices=0;
 	unsigned mb=0;        
int Rround=0;
	// max pattern size bit S
__global__ void searchb(char* data, char* pattern, int len_data,int len_substring, unsigned mb, unsigned* pos, unsigned *count_f)
{  

	// int i = threadIdx.x; //for 1 block
	
	//For all blocks
	int j,i =blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	const int numThreads = blockDim.x * gridDim.x;



	for (; i < len_data-mb+1; i+=numThreads  ) {

	    	
	// Detect the first matching character
	if (data[i] == pattern[0]) {
	
		
	// Loop through next keyword character
	for ( j=1; i < len_data && j<len_substring; j++) {
      if (data[i+j] != pattern[j])     break;     
	 }


	 if (j==len_substring) {
     // Store the first matching character to the result list
		 pos[tid]++;

		// atomicAdd(count_f, 1);
		  
	 }

	   
	}//end if matching
	
	}

}//end of Kernel

//
// Read in the given data file and hope it doesn't over the memory limits of the machine or that defined by 'DATA_SIZE'
//

	 FILE* f_b;
	 unsigned long long fileSize = 0;

	  char* data_b;  
	unsigned long long filesize(const char *filename)
	{
	FILE *f = fopen(filename,"rb");  /* open the file in read only */

		if (fseek(f,0,SEEK_END)==0) /* seek was successful */
			fileSize = ftell(f);
		fclose(f);
		return fileSize;
	}


	 int countR=0;

	char* readfile_b() { // read one gig    

	
			
			fread(data_b,sizeof(char),chunkSize,f_b);
	
		
    return data_b;
}

	 unsigned count_total_found(unsigned *arr, int n)
	{
		 int i;
		 unsigned c=0;
//		 printf("size %d \n",n);
		 for (i=0; i < n; i++)  {
			 
			   c += arr[i]; 
		 }
		 return c;
	}


int main(int argc, char** argv)
{
    printf("start\n");
	int cuda_device = 0; // device ID


	unsigned  arr_nb[MAX_CARDS];           // number of ints in the bit data set
	int j;

	int nStreams= MAX_STREAMS;
	 hipStream_t stream[MAX_CARDS]; 
	hipError_t result;

	//start Timer
	hipError_t error;   // capture returned error code
    hipEvent_t start_event, stop_event; // data structures to capture events in GPU
     float time_main_b, total_time_main_b=0.0;
	
	// Sanity checks
	{
	    // check the compute capability of the device A
       

		hipGetDeviceCount(&num_devices) ;
	    if(0==num_devices)
	    {
	        printf("your system does not have a CUDA capable A device\n");
	        return 1;
	    }
    	if( argc > 1 )
       		cuda_device = atoi( argv[1] );

	    // check if the command-line chosen device ID is within range, exit if not
	    if( cuda_device >= num_devices )
	    {
	        printf("choose device ID between 0 and %d\n", num_devices-1);
	        return 1;
	    }

    	hipSetDevice( cuda_device );

		if ( argc < 4 ) {
      		printf("Usage: StringmatchingGPU <device_number> <data_file_b> <string_pattern1-..99>\n");
      		return -1;
    	}
	} // end of safe checks

	//Cuda Device 
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	if( (deviceProp.major == 2) && (deviceProp.minor < 2)){ 
		printf("\n%s does not have compute capability 2.2 or later\n",deviceProp.name);}
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, cuda_device);
	printf("num SMs %d\n",numSMs);//print streaming multiprocessors
//--------------------------------------------------------	
  num_devices =2;
  for (int l=0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);
	nStreams = MAX_STREAMS;
		result = hipStreamCreate(&stream[l]);
 
  }

  error = hipGetLastError();
		if ( error ) { 	printf("Error caught: %s\n", hipGetErrorString( error ));}
		     
		
	//OpenFile
	if ((f_b = fopen(argv[2] , "r")) == NULL ) { printf("Error : read file\n"); return 0; }
	 
 
	long double total_diff2=0.0;	
        long double total_time_data = 0.0, total_time_pat =0.0, total_time_pos=0.0;

 
	
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	printf("pattern=");
	// copy str pattern to pattern array
	mb = 0;
	for (j=3; j < argc; j++)
	 {
		  pattern_arr[total_pattern] = (char*)malloc( (strlen(argv[j]) + 1) * sizeof(char) ); 
		 // count_found[total_pattern] = (int *) malloc( 2*sizeof(int));
		  //count_found[total_pattern]=0;
			
		  strcpy(pattern_arr[total_pattern],argv[j]);
		  printf(" %s ",pattern_arr[total_pattern]);
		   mb= (mb > strlen(pattern_arr[total_pattern])? mb : strlen(pattern_arr[total_pattern]));
		  total_pattern++;
		 
	 }
	printf("\n");
	//char* mainString_b;
	char* d_data_b = 0;
	unsigned *pos=0;
	unsigned *d_pos=0;
	 
	//Device's text
//	printf ("pointter size %ld \n",sizeof (unsigned*));
	
	// allocate D memory
	char* d_substr_b = 0;


	// read in the filename and string pattern to be searched
	 int alloc_size =(mb-1+chunkSize)*sizeof(char);
	char *arr_data_b[MAX_CARDS];
	int countc;

	unsigned *arr_pos[MAX_CARDS];
	unsigned *arr_d_pos[MAX_CARDS];
	char *arr_d_data_b[MAX_CARDS];
	char *arr_d_substr_b[MAX_CARDS];

	for (int l =0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);

	      hipHostMalloc((void**)&arr_data_b[l], alloc_size, hipHostMallocDefault) ; // host pinned
 
	    if (arr_data_b[l] == NULL) printf("alloc data_b error \n");  
	
	
	}
	 for (int l =0; l < num_devices; l++) {
			 
	     arr_pos[l] = (unsigned *)  calloc (TOTAL_THREADS, sizeof(unsigned));
		  
			if(arr_pos[l] == NULL) {printf("error alloc pos\n"); exit(-1);}
			
	 }


	for (int l =0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);
	
			hipMalloc((void**)&arr_d_pos[l],(TOTAL_THREADS)*sizeof(unsigned));//
			hipMalloc((void**)&d_count_f,(1)*sizeof(unsigned));//

     	 if (arr_d_pos[l] == NULL) {
			 printf("couldn't allocate d_pos\n"); exit(-1); }
		 
          hipMalloc((void**)&arr_d_data_b[l], (mb-1+chunkSize)*sizeof(char)) ;//
	  
	    if (arr_d_data_b[l] == NULL) {
			printf("couldn't allocate d_data_b\n"); exit(-1);  }
	  
		
	    hipMalloc((void**)&arr_d_substr_b[l], mb*sizeof(char));
	     if (arr_d_substr_b[l] == NULL) {
			 printf("couldn't allocate d_substr_b\n"); exit(-1);  }
	    
	}
	size_t cur_free, cur_total;

	printf("\n");
	for (int l =0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);
	
	      hipMemGetInfo(&cur_free,&cur_total); 

//    	   printf("free %ld KB of total %ld KB\n",cur_free/1024,cur_total/1024);
	}
	 
	 
int num_block;
	while ( !feof (f_b)) {
		num_block=0;
		for (int l =0; l < num_devices; l++) {  // looping read file chunk to devices
		 
		   countc=fread(arr_data_b[l],sizeof(char),chunkSize+mb-1,f_b);
		   if (countc <= 0 ) break;
			  
			arr_nb[l] =   ( unsigned ) countc/sizeof(char);
			if (!feof(f_b)) fseeko(f_b,-((long long)mb-1),SEEK_CUR);
//		  printf("read for card %d size %u \n",l, arr_nb[l]);
		  num_block++;
		}
 

	
	//Find 
		TOTAL_THREADS_PER_BLOCK = MAX_THREAD_PER_BLOCK ;  /**/

		

		//Print Block / Threads



		//H2D 
		t_data1= clock();
	// cudaMemcpy(d_data_b, data_b, nb, cudaMemcpyHostToDevice );//**maybe be Asynccopy

	 unsigned DATA_STEP  =1<<30;

		int sid = 0;
		 

		for (int l =0; l < num_devices; l++) {

//    int cset=l*2;
    hipSetDevice(l);

			if (l >= num_block)  break; // handle the case when  the numblock read is less than  numdevice
			hipMemcpyAsync( arr_d_data_b[l], arr_data_b[l],  arr_nb[l]*sizeof(char), hipMemcpyHostToDevice,stream[l]);
			printf("dev %d ",l);
		}	

	printf("\n");


		for (int l =0; l < num_devices; l++) {

//    int cset=l*2;
    hipSetDevice(l);
	
		     hipStreamSynchronize(stream[l]);
		}
		t_data2= clock();
	    long double diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
	
	    printf("timeCopyH2D-1 %Lf ms \n",diff2);Rround++;
		total_diff2 += diff2;
                total_time_data += diff2; 
		// using Kernel
		
		for (j=0; j < total_pattern; j++) {
			 	

//		   printf("after mem set pos pattern no. %d\n", j);
			t_data1= clock();
	    
			for (int l =0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);
			   
			   if (l >= num_block)  break;
			   hipMemcpy(arr_d_substr_b[l], pattern_arr[j], sizeof(char)*(strlen(pattern_arr[j])), hipMemcpyHostToDevice) ;
//			    printf("after copy to d_subs dev %d\n",l);
				hipMemcpy(arr_d_pos[l], arr_pos[l], (TOTAL_THREADS)*sizeof(unsigned), hipMemcpyHostToDevice) ; 
			}
			
		
			t_data2= clock();
		     diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
		        printf("timeCopyH2D-2 %Lf ms \n",diff2);Rround++;
                        total_diff2 += diff2;
			total_time_pat += diff2;
			 hipEventRecord(start_event, 0);
		
			
	       for (int l =0; l < num_devices; l++) {
			   
//    int cset=l*2;
    hipSetDevice(l);

//			   printf("Runing in kernel dev %d\n",l);
			   if (l >= num_block)  break;
			searchb <<<32*numSMs,TOTAL_THREADS,0,stream[l] >>>(arr_d_data_b[l], arr_d_substr_b[l],arr_nb[l],strlen(pattern_arr[j]),mb,arr_d_pos[l],&d_count_f[0]);
		   }
			hipEventRecord(stop_event, 0);
			hipEventSynchronize( stop_event );
			//Calculate time
			hipEventElapsedTime( &time_main_b, start_event, stop_event );
		//Getting Error 
		error = hipGetLastError();
		if ( error ) { 	printf("Error caught: %s\n", hipGetErrorString( error ));}
		     
		
	    for (int l =0; l < num_devices; l++) {
			  
//    int cset=l*2;
    hipSetDevice(l);

			t_data1 =clock();
			if (l >= num_block)  break;
			 //cudaMemcpyAsync(arr_pos[l], arr_d_pos[l], (arr_nb[l])*sizeof(bool), cudaMemcpyDeviceToHost,stream[l]) ; // result position
			hipMemcpy(arr_pos[l], arr_d_pos[l], (TOTAL_THREADS)*sizeof(unsigned), hipMemcpyDeviceToHost) ; 
			//cudaMemcpy(count_f, d_count_f,   sizeof(unsigned), cudaMemcpyDeviceToHost) ; 
	         error = hipGetLastError();
			if ( error ) { 	printf("Error caught ===: %s\n", hipGetErrorString( error ));}
		     
			 t_data2= clock();
		     diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
			 printf("\ntimeCopyH2D-3 %Lf ms ",diff2);Rround++;
			 total_diff2 += diff2;
			 total_time_pos +=diff2;
			//Print Time
			printf(" timeMainSearch %lf ms \n", time_main_b);
			total_time_main_b += time_main_b;
			//cudaHostGetDevicePointer((void **)&arr_pos[l], (void *)pos, 0);
			 
			
			// cleanup
		 }
		}
			 
		// stop timer
		
		
		
		//checkGpuMem(chunkSize);

		if (feof(f_b) || countc <=0 )
		 break;

		}//end while main stream
		//Free Substring


		
		printf("\ntimeMainSearch %lf ms ", total_time_main_b);
		printf("time copy H2D %Lf ms: total all = %Lf  ",total_diff2, total_diff2+total_time_main_b);

		for (int l=0; l < num_devices; l++)
		{
			 unsigned   t_f= count_total_found(arr_pos[l],TOTAL_THREADS);
			printf(" cur_found %u  \n", t_f);
			total_found += t_f;
		//printf("count_f %lu \n",count_f[0]);
		}

                printf("\ntimeCopyH2D %Lf ms ",total_diff2);
                printf("timeMainSearch %lf ms ", total_time_main_b);
                printf("totalAll = %Lf ", total_diff2+total_time_main_b);
                printf("Found %d ",total_found);
                printf("Round %d \n", Rround);

printf("\ntotal_time_data %Lf ms total_time_pat %Lf ms " , total_time_data, total_time_pat);
printf(" total_time_pos %Lf ms TotalH2D %Lf ms ", total_time_pos, total_time_data+total_time_pat+total_time_pos);

		
		

		//Free Input
		   
		 
	 
	for (int l =0; l < num_devices; l++) {
		

//    int cset=l*2;
    hipSetDevice(l);
	
		hipFree(arr_d_data_b[l]); 
		hipFree(arr_d_pos[l]); 
		hipFree(arr_d_substr_b[l]);
		//cudaFree(d_pos);
	}
		hipEventDestroy( start_event ); 
		hipEventDestroy( stop_event ); 

			 
	for (int l =0; l < num_devices; l++) {
	 
		free(arr_pos[l]);
		//free(arr_data_b[l]);
		hipHostFree(arr_data_b[l]);
		//    cudaFreeHost(arr_pos[l]);
	}
		for (j=0; j < total_pattern; j++) { 
			free(pattern_arr[j]);
		  }
		//Close Input File
		fclose(f_b);
	for (int l =0; l < num_devices; l++) {
//    int cset=l*2;
    hipSetDevice(l);
		
		//for (int i = 0; i < nStreams; ++i)  {
		 result = hipStreamDestroy(stream[l]);
 
	//}
	}
	hipFree(d_count_f);

		printf("\nEnd");
		return 0;

}
