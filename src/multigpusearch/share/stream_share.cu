#define _LFS_LARGEFILE          1
#define _LFS64_LARGEFILE        1
#define _LFS64_STDIO		1
#define _LARGEFILE64_SOURCE    	1
#define _FILE_OFFSET_BITS 64

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define MAX_STREAMS 4
#define MAX_CARDS 2

#define BLOCK_SIZE 1014
#define MAX_THREAD_PER_BLOCK 1014

long long unsigned total_sub,total_data;
clock_t t_data1,t_data2;

//const unsigned long long chunkSize =1<<25;

unsigned long work_per_thread = 100;

#define MAX 1000

#define PATTERNSIZE 32
#define MAX_TOTAL_PATTERN 16

__constant__ char cpattern[sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN];


char *pattern_arr[MAX];
long long unsigned *count_found[MAX];

int total_pattern;
long long  unsigned total_found;
int TOTAL_THREADS_PER_BLOCK ;
int num_devices=0;
unsigned mb=0;        
int Rround=0;
int max_str;

#ifdef CHUNK2G
unsigned long long chunkSize =0x80000000; //2G
#elif CHUNK3G
unsigned long long chunkSize =0xc0000000; //3G
#elif CHUNK4G
 unsigned long long chunkSize =0x100000000; //4G
#elif CHUNK5G
unsigned long long chunkSize =0x140000000; //5G
#elif CHUNK1G
 unsigned long long chunkSize =0x40000000; //1G
#elif CHUNK6G
unsigned long long chunkSize =0x180000000; //6G
#else
unsigned long long chunkSize =0x79999999; //1.99G default
#endif

__global__ void searchb(
char* data, 
unsigned long long len_data,
unsigned mb, 
short* pos,
int p_count)
{   

	// int i = threadIdx.x; //for 1 block
	unsigned mycount=0;
	//For all blocks
	unsigned long long  j,i =blockIdx.x * blockDim.x + threadIdx.x;
	char *substring;
	const int numThreads = blockDim.x * gridDim.x;
	int p;


	 __syncthreads();

	int  found=0;
       printf("kernel\n");

	for (; i < len_data-mb+1; i+=numThreads  ) {

	  found=0;  	
	  for (p=0; p < p_count; p++ ) {

	     substring=(char*) (cpattern+p*PATTERNSIZE);
	

           found=1;
	// Detect the first matching character
	   if (data[i] == *substring) {
              printf("first match\n");	
		
	// Loop through next character
	     char *cur=(substring+1);
	     for ( j=1; *cur !=0 && j<PATTERNSIZE ; j++,cur++) {
        	if (data[i+j] != *cur)     
		{
		  found=0;		
		  break; 
  	 	 }
	     }
	  }	
	  if (found) {
	     pos[i]++;      
	     mycount++; 
	  }
        } // end p
      } //end numthread

}//end of Kernel

__global__ void searchb_all(char* data,  unsigned long long len_data, unsigned mb, short* pos, int p_count)
{

        unsigned long long mycount=0;
        //For all blocks
        unsigned long long j,i =blockIdx.x * blockDim.x + threadIdx.x;

        const int numThreads = blockDim.x * gridDim.x;
	char found,k;
	char *pattern;

   for (; i < len_data-mb+1; i+=numThreads  ) {

        for(k=0; k < p_count; k++) {
	    found=0;
	    pattern = (cpattern+k*PATTERNSIZE);	

           if (data[i] == pattern[0]) {
	      found=1;


            for ( j=1; i+j < len_data && pattern[j] != '\0' && j<PATTERNSIZE; j++) {
             if (data[i+j] != pattern[j])     {found=0; break;}
            }

             if (found) {
                 pos[i] += 1; 
                 mycount++;
             }
          }//end if matching
       } // end for p_count

   }

}//end of Kernel


__global__ void searchb_all_shared(char* data, char* allpattern, unsigned long long len_data, unsigned mb, short* pos, int p_count)
{

        unsigned long long mycount=0;

	unsigned long long j,i =blockIdx.x * blockDim.x + threadIdx.x;

        const int numThreads = blockDim.x * gridDim.x;
        char found,k;
        char *pattern=allpattern;
	__shared__ char allpattern_s [MAX_TOTAL_PATTERN*PATTERNSIZE];

	if (threadIdx.x < MAX_TOTAL_PATTERN*PATTERNSIZE)
         	allpattern_s[threadIdx.x]= allpattern[threadIdx.x];

	 __syncthreads();

     for (; i < len_data-mb+1; i+=numThreads  ) {

        for(k=0; k < p_count; k++) {
            found=0;
            pattern = (allpattern_s+k*PATTERNSIZE);

           if (data[i] == pattern[0]) {
              found=1;

            for ( j=1; i+j < len_data && pattern[j] != '\0' && j<PATTERNSIZE; j++) {
             if (data[i+j] != pattern[j])     {found=0; break;}
            }


             if (found) {
                 pos[i] += 1;
                 mycount++;
             }

          }//end if matching
       } // end for p_count

   }

}//end of Kernel

	 FILE* f_b;
	 unsigned long long fileSize = 0;

	  char* data_b;  
	unsigned long long filesize(const char *filename)
	{
	FILE *f = fopen(filename,"rb");  /* open the file in read only */

		if (fseek(f,0,SEEK_END)==0) /* seek was successful */
			fileSize = ftell(f);
		fclose(f);
		printf("fileSize = %llu", fileSize);
		return fileSize;
	}


	 int countR=0;

	char* readfile_b() { // read one gig    
	fread(data_b,sizeof(char),chunkSize,f_b);
    return data_b;
}



	unsigned long long count_total_found(short *arr,  unsigned long long   n)
	{
		 unsigned long long i;
		 unsigned long long c=0;
		 printf(" size %lld ",n);
		 for (i=0; i < n; i++)  {
			 //printf("i%ld arr[i] %d\n", i, arr[i]);  
			 if (arr[i] >0 ) { 
 			    c += (unsigned long long) arr[i]; //("here:");
			 }
		 }
		 return c;
	}


int main(int argc, char** argv)
{
    printf("start chunksize %ld \n",chunkSize);
	int cuda_device = 0; // device ID


	unsigned long long arr_nb[MAX_CARDS];          
	int j;

	int nStreams= MAX_STREAMS;
	 hipStream_t stream[MAX_CARDS]; 
	hipError_t result;


	hipError_t error;   // capture returned error code
    hipEvent_t start_event, stop_event; // data structures to capture events in GPU
     float time_main_b, total_time_main_b=0.0;
	

	{
	hipGetDeviceCount(&num_devices) ;
	    if(0==num_devices)
	    {
	        printf("your system does not have a CUDA capable A device\n");
	        return 1;
	    }
    	 
		if ( argc < 3 ) {
      		printf("Usage:  %s   <data_file_b> <string_substring1-..99>\n",argv[0]);
      		return -1;
    	}
	} 
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	if( (deviceProp.major == 2) && (deviceProp.minor < 2)){ 
		printf("\n%s does not have compute capability 2.2 or later\n",deviceProp.name);}
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, cuda_device);
	printf("num SMs %d\n",numSMs);//print streaming multiprocessors
	
  for (int l=0; l < num_devices; l++) {
    hipSetDevice(l);
	nStreams = MAX_STREAMS;
	result = hipStreamCreate(&stream[l]);
 
  }

  	error = hipGetLastError();
	if ( error ) { 	
		printf("Error caught-1: %s\n", hipGetErrorString( error ));
	}
		     
	if ((f_b = fopen(argv[1] , "r")) == NULL ) { printf("Error : read file %s\n",argv[1]); return 0; }
	 
 
	long double total_diff2=0.0,diff2=0;	
    long double total_time_data = 0.0, total_time_pat =0.0, total_time_pos=0.0;
 
   
	
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	printf("Constant memory substring=");

	mb = 0;
 	char *pattern= (char *) malloc(sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN);
	if (pattern == NULL) 
	printf("error alloc whole patterns\n");
	memset(pattern,0,sizeof(char)*PATTERNSIZE*MAX_TOTAL_PATTERN); 
         pattern_arr[0] =  pattern;
	for (j=2; j < argc; j++)
	 {
		  pattern_arr[total_pattern] = pattern+(PATTERNSIZE*total_pattern);
		  memcpy(pattern_arr[total_pattern], argv[j], sizeof(char)*(strlen(argv[j])+1));
		  printf(" %s ",pattern_arr[total_pattern]);
		   mb= (mb > strlen(pattern_arr[total_pattern])? mb : strlen(pattern_arr[total_pattern]));
		  total_pattern++; 
		 
	 }
 	max_str = mb;
	printf("\n total_pattern =%d \n", total_pattern);

	char* d_data_b = 0;
	short *pos=NULL;
	short *d_pos=NULL;

	char* d_substr_b = 0;


	 unsigned long long  alloc_size =(mb-1+chunkSize)*sizeof(char);
	char *arr_data_b[MAX_CARDS];
	unsigned long long  countc;

	short *arr_pos[MAX_CARDS];
	short *arr_d_pos[MAX_CARDS];
	char *arr_d_data_b[MAX_CARDS];
	char *arr_d_substr_b[MAX_CARDS];

	for (int l =0; l < num_devices; l++) {
		  hipSetDevice(l);
	      hipMallocManaged(&arr_data_b[l], alloc_size) ; // unified memory
 
	    if (arr_data_b[l] == NULL) printf("hipMallocManaged data_b error \n");  
	
	
	}
	 for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
			arr_pos[l] = (short *) malloc (((mb-1)+chunkSize)*sizeof(short));
		  
			if(arr_pos[l] == NULL) {printf("error alloc pos\n"); exit(-1);}
			
	 }


	for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
	
	        hipMalloc((void**)&arr_d_pos[l],(mb-1+chunkSize)*sizeof(short));//

     	    if (arr_d_pos[l] == NULL)
	    	  printf("couldn't allocate d_pos device %d\n",l);

    if ( error ) {  printf("Error caught-hipMemcpy d_substr_b: %s %d\n", hipGetErrorString( error ),error);}  

		for (j=0; j < total_pattern; j++)
	           printf("copy arr_d_sub %s \n",pattern_arr[j]);

            error =hipMemcpyToSymbol    (  HIP_SYMBOL(cpattern),  pattern_arr[0],  
             PATTERNSIZE*MAX_TOTAL_PATTERN*sizeof(char) );

               if ( error ) {  printf("Error caught-hipMemcpy arr_d_substr_b: %s %d device %d\n", hipGetErrorString( error ),error,l);}  
	}

	size_t cur_free, cur_total;

	printf("\n");
	for (int l =0; l < num_devices; l++) {
			hipSetDevice(l);
	
	      hipMemGetInfo(&cur_free,&cur_total); 

    	   printf("device %d: free %ld KB of total %ld KB\n",l,cur_free/1024,cur_total/1024);
	}
	 
	 
	int num_block;
	while ( !feof (f_b)) {
		num_block=0;
		for (int l =0; l < num_devices; l++) {  // looping read file chunk to devices
		  
		   countc=fread(arr_data_b[l],sizeof(char),chunkSize+mb-1,f_b);
		   if (countc <= 0 ) break;
			  
			arr_nb[l] =   ( unsigned long long ) (countc/sizeof(char));
			if (!feof(f_b)) fseeko(f_b,-((unsigned long long)mb-1),SEEK_CUR);
		  printf("read for card %d size %u \n",l, arr_nb[l]);
		  num_block++;
		}
 
		TOTAL_THREADS_PER_BLOCK = MAX_THREAD_PER_BLOCK ;  /**/

		Rround++;
		// using Kernel
		
		 	
		for (int l =0; l < num_devices; l++) {
			   hipSetDevice(l);
			   memset(arr_pos[l],(short) 0,arr_nb[l]*sizeof(short));
                           //for (unsigned long long d=0; d < arr_nb[l]-mb+1; d++)
                            //  if (arr_pos[l][d] != 0) {   printf("not clear arrpos\n"); }
			   hipMemset(arr_d_pos[l],(short)0,sizeof(short)*arr_nb[l]);
		}

                error = hipGetLastError();
                        if ( error ) {  printf("Error caught-2- memset d_pos: %s\n", hipGetErrorString( error ));}
	         printf("after mem set pos pattern . \n" );
			 
		// stop timer
		
		
	        hipEventRecord(start_event, 0);


               for (int l =0; l < num_devices; l++) {
                           hipSetDevice(l);
                         printf("RunK>Dev %d\n ",l);
                           if (l >= num_block)  break;
	 
			searchb_all_shared <<<16*numSMs,128,0,stream[l] >>>(arr_d_data_b[l], arr_d_substr_b[l],arr_nb[l],mb,arr_d_pos[l],total_pattern);
	 
                error = hipGetLastError();
                if ( error ) {  printf("Error caught-1: %s\n", hipGetErrorString( error ));}
                   }	
			 hipEventRecord(stop_event, 0);
                        hipEventSynchronize( stop_event );

                        hipEventElapsedTime( &time_main_b, start_event, stop_event );

                error = hipGetLastError();
                if ( error ) {  printf("Error caught-1: %s\n", hipGetErrorString( error ));}

                for (int l =0; l < num_devices; l++) {
                          hipSetDevice(l);
                        t_data1 =clock();
                        if (l >= num_block)  break;
                        hipMemcpy(arr_pos[l], arr_d_pos[l], (arr_nb[l])*sizeof(short), hipMemcpyDeviceToHost) ;
                         error = hipGetLastError();
                        if ( error ) {  printf("Error caught-2: %s\n", hipGetErrorString( error ));}

                         t_data2= clock();
                         diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) *1000;
                         printf("timeCopyH2D-3 %Lf ms ",diff2);
                         Rround++;
                         total_diff2 += diff2;
                         total_time_pos += diff2;

                        printf(" timeMainSearch %lf ms ", time_main_b);
                        total_time_main_b += time_main_b;

                         unsigned long long  t_f= count_total_found(arr_pos[l],arr_nb[l]-mb+1);
                         printf ("nb %llu : \n ",arr_nb[l]);

                        printf("current_found %llu  \n", t_f);
                        total_found += t_f;
                 }

		if (feof(f_b) || countc <=0 )
		 break;
		}
		
                printf("\ntimeCopyH2D %Lf ms ",total_diff2);
		printf("time main search %lf ms ", total_time_main_b);
                printf("totalAll = %Lf ", total_diff2+total_time_main_b);
		printf("Found %lu ",total_found);
                printf("Round %d \n", Rround);

		printf("\ntotal_time_data %Lf ms total_time_pat %Lf ms " , total_time_data, total_time_pat);
                printf(" total_time_pos %Lf ms TotalH2D %Lf ms ", total_time_pos, total_time_data+total_time_pat+total_time_pos);

	
			 
	for (int l =0; l < num_devices; l++) {
		hipSetDevice(l);
		hipFree(arr_d_pos[l]); 
	 
	}
		hipEventDestroy( start_event ); 
		hipEventDestroy( stop_event ); 

			 
	for (int l =0; l < num_devices; l++) {
		free(arr_pos[l]);
		hipFree(arr_data_b[l]);
	}
 
	 free(pattern);

		fclose(f_b);
    	for (int l =0; l < num_devices; l++) {
		hipSetDevice(l);
		 result = hipStreamDestroy(stream[l]);
 
	}
		printf("\nEnd");
		return 0;

}