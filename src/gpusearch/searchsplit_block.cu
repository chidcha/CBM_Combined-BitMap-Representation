#include "hip/hip_runtime.h"
#define _LFS_LARGEFILE          1
#define _LFS64_LARGEFILE        1
#define _LFS64_STDIO			1
#define _LARGEFILE64_SOURCE    	1

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <time.h>

///////////////////////////////////////////////////////////////
//set block size
///////////////////////////////////////////////////////////////
#ifdef BLOCK_SIZE32
#define BLOCK_SIZE 32
#elif BLOCK_SIZE64
#define BLOCK_SIZE 64
#elif BLOCK_SIZE128
#define BLOCK_SIZE 128
#elif BLOCK_SIZE256
#define BLOCK_SIZE 256
#elif BLOCK_SIZE512
#define BLOCK_SIZE 512
#elif BLOCK_SIZE512
#define BLOCK_SIZE 512
#elif BLOCK_SIZE1024
#define BLOCK_SIZE 1024
#else
#define BLOCK_SIZE 32
#endif


#define MAX_THREAD_PER_BLOCK 1014
long long unsigned total_sub, total_data;
clock_t t_sub1, t_sub2, t_data1, t_data2;

///////////////////////////////////////////////////////////////
//set chunk size
///////////////////////////////////////////////////////////////
#ifdef CHUNK2G
unsigned long long chunkSize = 0x80000000; //2G
#elif CHUNK3G
unsigned long long chunkSize = 0xc0000000; //3G
#elif CHUNK4G
unsigned long long chunkSize = 0x100000000; //4G
#elif CHUNK5G
unsigned long long chunkSize = 0x140000000; //5G
#elif CHUNK1G
unsigned long long chunkSize = 0x40000000; //1G
#elif CHUNK6G
unsigned long long chunkSize = 0x180000000; //6G
#elif CHUNK1M
unsigned long long chunkSize = 0x100000;//1MB
#elif CHUNK32M
unsigned long long chunkSize = 0x2000000;//32MB
#elif CHUNK256M
unsigned long long chunkSize =0x10000000;//256MB
#elif CHUNK32KB
unsigned long long chunkSize = 0x8000;//32KB
#else
unsigned long long chunkSize =0x8000000;//128MB
#endif


unsigned long work_per_thread = 100;
#define MAX 100
char *pattern_arr[MAX];
int *count_found[MAX];
int total_pattern;
long unsigned total_found;
int TOTAL_THREADS_PER_BLOCK;



int Rround = 0;

__global__ void searchb(char* data, char* pattern, int len_data, int len_substring, bool*pos, unsigned long work_size)//, int* results)
{

	// int i = threadIdx.x; //for 1 block

	//For all blocks
	int j, i = blockIdx.x * blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * gridDim.x;

	for (; i < len_data; i += numThreads) {

		if (data[i] == pattern[0]) {

			for (j = 1; i + j < len_data && j<len_substring; j++) {
				if (data[i + j] != pattern[j])    { //yes = 0; 
					break; 
				}
			}

			if (j == len_substring) {
				pos[i] = true;
			}
			else  pos[i] = false; //end if marking position


		}//end if matching

	}

}//end of Kernel

//
// Read in the given data file and hope it doesn't over the memory limits of the machine or that defined by 'DATA_SIZE'
//

FILE* f_b = NULL;
FILE* f_t = NULL;
FILE*pFile = NULL;
unsigned long long fileSize = 0;


size_t currByte = 0;


unsigned long long filesize(const char *filename)
{
	FILE *f = fopen(filename, "rb");  /* open the file in read only */
#ifdef __linux__
	if (fseeko(f, 0, SEEK_END) == 0) /* seek was successful */
		fileSize = ftell(f);
	fclose(f);
#elif _WIN32
	if (_fseeki64(f, 0, SEEK_END) == 0) /* seek was successful */
		fileSize = ftell(f);
	fclose(f);

#endif
	printf("fileSize = %llu", fileSize);
	return fileSize;
}


int countR = 0;



long unsigned count_total_found(bool *arr, int n)
{
	int i;
	long unsigned c = 0;
	for (i = 0; i < n; i++)
	{
		if (arr[i]) {
			c++;
			//printf("%d,%u\n",i, c);//position, order
			printf("%d\n", i);//position
		}
	}
	return c;
}


void checkGpuMem(unsigned long long size)
{
	double free_m, total_m, used_m, mem_used, temp1, here1;
	size_t free_t, total_t, temp;
	unsigned int mem, rana, here;

	hipMemGetInfo(&free_t, &total_t);
	free_m = (unsigned int)free_t / 1048576.0;
	total_m = (unsigned int)total_t / 1048576.0;
	temp = (unsigned int)size / 1048576.0;
	used_m = total_m - free_m;
	here1 = free_t - (unsigned int)size / 1048576.0;
	mem = free_m - temp;
	rana = temp - free_m;
	printf(" checkGPU mem %lu %lf %ud %lf %lf %i %i %lf\n", free_t, free_m, (unsigned)total_t, total_m, used_m, mem, rana, here1);
	//printf ( "mem free %d .... %f MB mem \ntotal %d....%f MB mem \nused %f MB\n",free_t,free_m,total_t,total_m,used_m);
	//printf(" mem free after array %i MB\n",mem);
	//printf(" negative mem free after array %i MB\n",rana);
	//printf(" bytes mem free after array %i MB\n",here1);

}


void printDevProp(hipDeviceProp_t devProp)
{
	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
	printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %lu\n", devProp.totalConstMem);
	printf("Texture alignment:             %lu\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}



//void print_shifts(int *iptr, int strlen) {
//	for (unsigned int i = 0; i < strlen; i++) {
//		if (iptr[i] == 1)
//			printf("%d\n", i);
//	}
//}

//char* readfile(const char* filename) {
//	FILE* f;
//	char* data = (char*)malloc(1181741 * sizeof(char));
//
//	if ((f = fopen(filename, "r")) != NULL) {
//		// read in the entire file and store into memory
//		// hopefully it doesn't exhause the entire RAM on
//		// the machine or defy the limits as defined by DATA_SIZE
//		fscanf(f, "%s", data);
//	}
//	fclose(f);
//	return data;
//}

int main(int argc, char** argv)
{
	printf("start\n");
	int cuda_device = 0; // device ID
	long dposSize = 0;
	int mb = 0;           // pattern size bit S
	int nb = 0;           // number of ints in the bit data set
	int j, k;


	//int increasestep=1;

	//start Timer
	hipError_t error;   // capture returned error code
	hipEvent_t start_event, stop_event; // data structures to capture events in GPU
	float time_main_b;
	double total_time_main_b = 0.0;

	// Sanity checks
	{
		// check the compute capability of the device A
		int num_devices = 0;

		hipGetDeviceCount(&num_devices);
		if (0 == num_devices)
		{
			printf("your system does not have a CUDA capable A device\n");
			return 1;
		}
		//if (argc > 1)
			cuda_device = atoi("0");

		// check if the command-line chosen device ID is within range, exit if not
		if (cuda_device >= num_devices)
		{
			printf("choose device ID between 0 and %d\n", num_devices - 1);
			return 1;
		}

		hipSetDevice(cuda_device);

		//if (argc < 4) {
		//	printf("Usage: StringmatchingGPU <device_number> <data_file_b> <string_pattern1-..99>\n");
		//	return -1;
		//}
	} // end of safe checks

	//Cuda Device 
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cuda_device);
	//printDevProp(deviceProp);
	if ((deviceProp.major == 2) && (deviceProp.minor < 2)){
		printf("\n%s does not have compute capability 2.2 or later\n", deviceProp.name);
	}
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, cuda_device);
	printf(" num SMs %d\n", numSMs);
	// printf(" %s ", deviceProp.name );
	//  printf(" %d.%d %d ", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	//char bitfilename[60];
	char uriIdx[3][2];

	//OpenFile
	
//	if ((f_b = fopen("I:\\Compress\\swdf_2012_11_28_b.txt", "r")) == NULL) { printf("Error : read file b\n"); return 0; }

	if ((f_b = fopen("/data/noo/data/compress/freebase10M_b.txt", "r")) == NULL) { printf("Error : read file b\n"); return 0; }

//"/data/noo/data/compress/freebase10M_b.txt"
//"/data/noo/data/compress/freebase10M_t.txt"


	//filesize(argv[2]);
	//if ((f_t = fopen("I:\\Compress\\swdf_2012_11_28_t.txt", "r")) == NULL) { printf("Error : read file t\n"); return 0; }
	//unsigned long chunkSize = 1073741824;
	unsigned long long currSize = fileSize;
	long double total_diff2 = 0.0;
	long double total_time_data = 0.0, total_time_pat = 0.0, total_time_pos = 0.0;



	while (currSize>chunkSize){
		currSize = (unsigned long)(currSize - chunkSize);
		//printf("\nround |");
		countR++;
	}

	strcpy(uriIdx[0], "11");
	//strcpy(uriIdx[1], "10");
	//strcpy(uriIdx[2], "0");


	//Substring
	char* subString_b = (char*)malloc((strlen(uriIdx[0]) + 1) * sizeof(char));
	strcpy(subString_b, uriIdx[0]);



	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	// copy str pattern to pattern array
	mb = 0;
	for (j = 0; j < 1; j++)
	{
		pattern_arr[total_pattern] = (char*)malloc((strlen(uriIdx[j]) + 1) * sizeof(char));
		count_found[total_pattern] = (int *)malloc(2 * sizeof(int));
		count_found[total_pattern] = 0;

		strcpy(pattern_arr[total_pattern], uriIdx[j]);
		printf("pattern= %s \n", pattern_arr[total_pattern]);
		mb = (mb > strlen(pattern_arr[total_pattern]) ? mb : strlen(pattern_arr[total_pattern]));

		total_pattern++;
	}

	char* mainString_b;
	char* d_data_b = 0, *data_b;
	bool* pos = 0;
	bool* d_pos = 0;
	//Device's text


	// allocate D memory
	char* d_substr_b = 0;

	// read in the filename and string pattern to be searched

	int alloc_size = (chunkSize + mb - 1)*sizeof(char);
	int countc;
	unsigned int cur_size, my_size;
	char *cur_p, *next_p;

	data_b = (char *)malloc((chunkSize + mb - 1)*sizeof(char));
	pos = (bool *)malloc((chunkSize + mb - 1)*sizeof(bool));
	hipMalloc((void**)&d_pos, (chunkSize + mb - 1)*sizeof(bool));//
	if (d_pos == NULL)
		printf("couldn't allocate d_pos\n");
	dposSize = dposSize + (long)pos;

	hipMalloc((void**)&d_data_b, alloc_size);//
	if (d_data_b == NULL)
		printf("couldn't allocate d_data_b\n");
	hipMalloc((void**)&d_substr_b, (strlen(subString_b))*sizeof(char));

	
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//char* mainString = readfile(argv[2]);
	//int*  d_finalres = 0;
	//int* finalres = (int*)malloc((strlen(mainString))*sizeof(int));
	//
	//hipMalloc((void**)&d_finalres, (strlen(mainString))*sizeof(int));
	//hipMemset(d_finalres, 0, sizeof(int)*strlen(mainString));
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	size_t cur_free, cur_total;



	printf("\n");

	hipMemGetInfo(&cur_free, &cur_total);

	printf("%ld KB free of total %ld KB\n", cur_free / 1024, cur_total / 1024);


	while ((countc = fread(data_b, sizeof(char), (chunkSize + mb - 1), f_b))>0){


		mainString_b = data_b;
		nb = (int)countc / sizeof(char);
		nb = nb - (mb - 1);
		printf("size read (byte) %d ", nb);


		TOTAL_THREADS_PER_BLOCK = MAX_THREAD_PER_BLOCK;  /**/


		dim3 threadsPerBlocks(TOTAL_THREADS_PER_BLOCK, 1);
		dim3 numBlocks((int)ceil((double)nb / TOTAL_THREADS_PER_BLOCK), 1);

		work_per_thread = (unsigned long)(ceil((double)BLOCK_SIZE / TOTAL_THREADS_PER_BLOCK));

		//Print Block / Threads

		printf("numblock %d  thread perblock %d work perThread %ld\n", numBlocks.x, threadsPerBlocks.x, work_per_thread);

		if (work_per_thread <= 0) work_per_thread = 1;


		//H2D 
		t_data1 = clock();
		hipMemcpy(d_data_b, data_b, (nb + (mb - 1)), hipMemcpyHostToDevice);//**maybe be Asynccopy
		t_data2 = clock();
		long double diff2 = (((double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) * 1000;

		printf("timeCopyH2D-1 %Lf ms \n", diff2);
		Rround++;
		total_diff2 += diff2;
		total_time_data += diff2;

		// start timer!

		// using Kernel
		for (j = 0; j < total_pattern; j++) {
			//mb = strlen(pattern_arr[j]);


			//pos[0] = -1;
			memset(pos, false, nb);
			hipMemset(d_pos, false, nb);
			t_data1 = clock();
			hipMemcpy(d_substr_b, pattern_arr[j], sizeof(char)*(strlen(pattern_arr[j])), hipMemcpyHostToDevice);

			t_data2 = clock();
			diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) * 1000;
			printf("timeCopyH2D-2 %Lf ms \n", diff2);
			Rround++;
			total_diff2 += diff2;
			total_time_pat += diff2;


			hipEventRecord(start_event, 0);
			///////////////////////////////////////////////////////////////////////////////////////////////////////
			//Call kernel
			///////////////////////////////////////////////////////////////////////////////////////////////////////
			searchb << <BLOCK_SIZE,1024>> >(d_data_b, d_substr_b, nb, strlen(pattern_arr[j]), d_pos, work_per_thread);
			

			hipEventRecord(stop_event, 0);
			hipEventSynchronize(stop_event);
			//Calculate time
			hipEventElapsedTime(&time_main_b, start_event, stop_event);
			//Getting Error 
			error = hipGetLastError();
			if (error) { printf("Error caught: %s\n", hipGetErrorString(error)); }
			t_data1 = clock();
			hipMemcpy(pos, d_pos, nb, hipMemcpyDeviceToHost); // result position
			//hipMemcpy(finalres, d_finalres, (strlen(mainString))*sizeof(int), hipMemcpyDeviceToHost);
			t_data2 = clock();
			diff2 = (((long double)t_data2 - (double)t_data1) / CLOCKS_PER_SEC) * 1000;
			printf("timeCopyH2D-3 %Lf ms \n", diff2);
			Rround++;
			total_diff2 += diff2;
			total_time_pos += diff2;

			//Print Time
			printf("timeMainSearch %f ms ", time_main_b);
			total_time_main_b += time_main_b;
			int t_f = count_total_found(pos, nb);
			printf(" cur_found %d  \n", t_f);
			total_found += t_f;

			//printf("-------------------------------\n");
			//print_shifts(finalres, strlen(mainString) + 1);
			//printf("-------------------------------\n");
			// cleanup

			// hipMemcpy(pos, d_pos, sizeof(int)*2, hipMemcpyDeviceToHost) ;
			//if (pos[0] != -1)
			//count_found[j]++;
		}
		// stop timer



		//		checkGpuMem(chunkSize);

#ifdef __linux__
		if (!feof(f_b)) fseeko(f_b, -((long long)mb - 1), SEEK_CUR);
		else break;
#elif _WIN32
		if (!feof(f_b)) _fseeki64(f_b, -((long long)mb - 1), SEEK_CUR);
		else break;
#endif


	}//end while main stream

	//Free Substring

	hipFree(d_substr_b);
	free(subString_b);

	printf("\ntimeCopyH2D %Lf ms ", total_diff2);
	printf("timeMainSearch %lf ms ", total_time_main_b);
	printf("totalAll = %Lf ", total_diff2 + total_time_main_b);
	printf("Found %lu ", total_found);
	printf("Round %d \n", Rround);

	printf("\ntotal_time_data %Lf ms total_time_pat %Lf ms ", total_time_data, total_time_pat);
	printf(" total_time_pos %Lf ms TotalH2D %Lf ms ", total_time_pos, total_time_data + total_time_pat + total_time_pos);
	printf("\n dposSize %ld bool %zu\n", dposSize, sizeof(bool));

	//} //end for receive Pattern


	//Free Input

	//free(mainString_b);
	free(data_b);

	hipFree(d_data_b);
	hipFree(d_pos);

	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

	//               printf("\npos");
	free(pos);

	for (j = 0; j < total_pattern; j++)
	{
		free(pattern_arr[j]);
		free(count_found[j]);
	}
	//Close Input File
	/*hipFree(d_finalres);
	free(finalres);*/

	//                printf("\nfclose end");
	fclose(f_b);

	printf("\nEnd");
	return 0;

}
